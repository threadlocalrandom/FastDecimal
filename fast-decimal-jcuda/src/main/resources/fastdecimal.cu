
#include <hip/hip_runtime.h>
/**
 * CUDA kernels for FastDecimal operations.
 * These kernels implement basic arithmetic operations for arrays of scaled decimal values.
 */

/**
 * Adds two arrays of scaled decimal values element-wise.
 *
 * @param a The first input array
 * @param b The second input array
 * @param result The output array to store the results
 * @param n The length of the arrays
 */
extern "C" __global__ void add_kernel(long* a, long* b, long* result, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i];
    }
}

/**
 * Subtracts one array of scaled decimal values from another element-wise.
 *
 * @param a The first input array (minuend)
 * @param b The second input array (subtrahend)
 * @param result The output array to store the results
 * @param n The length of the arrays
 */
extern "C" __global__ void subtract_kernel(long* a, long* b, long* result, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = a[i] - b[i];
    }
}

/**
 * Multiplies two arrays of scaled decimal values element-wise.
 * The result is scaled by dividing by the scale factor to maintain the correct scale.
 *
 * @param a The first input array
 * @param b The second input array
 * @param result The output array to store the results
 * @param n The length of the arrays
 * @param scale_factor The scale factor used for internal representation
 */
extern "C" __global__ void multiply_kernel(long* a, long* b, long* result, int n, long scale_factor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = (a[i] * b[i]) / scale_factor;
    }
}

/**
 * Divides one array of scaled decimal values by another element-wise.
 * The dividend is scaled up by multiplying by the scale factor to maintain precision.
 * HALF_UP rounding is applied to match the behavior of FastDecimal.divide().
 *
 * @param a The first input array (dividend)
 * @param b The second input array (divisor)
 * @param result The output array to store the results
 * @param n The length of the arrays
 * @param scale_factor The scale factor used for internal representation
 */
extern "C" __global__ void divide_kernel(long* a, long* b, long* result, int n, long scale_factor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (b[i] == 0) {
            // Set a special value to indicate division by zero
            // This will be checked later in the Java code
            result[i] = 9223372036854775807L; // Long.MAX_VALUE
        } else {
            // Scale up the dividend to maintain precision
            long dividend = a[i] * scale_factor;
            
            // Calculate quotient and remainder
            long quotient = dividend / b[i];
            long remainder = dividend % b[i];
            
            // Apply HALF_UP rounding
            long halfDivisor = (b[i] < 0 ? -b[i] : b[i]) / 2;
            if ((remainder < 0 ? -remainder : remainder) >= halfDivisor) {
                result[i] = quotient + (a[i] >= 0 ? 1 : -1);
            } else {
                result[i] = quotient;
            }
        }
    }
}